#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <string.h>

#define MAX_LINE_LENGTH 1024
#define G 6.67E-11

const int nthreads = 4;

__device__ void update_points(float *fx, float* fy, float *masses, float *array_x, float *array_y,
 float *v_x, float *v_y, int n, float delta_t) 
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //printf("update points idx: %d\n", i);

    array_x[i] += v_x[i] * delta_t;
    array_y[i] += v_y[i] * delta_t;
    v_x[i] += (fx[i] / masses[i]) * delta_t;
    v_y[i] += (fy[i] / masses[i]) * delta_t;
}

__global__ void calculate_force(float *fx, float* fy, float *masses, float *array_x, float *array_y,
 float *v_x, float *v_y, int n, float delta_t) 
{
    int my_idx = blockDim.x * blockIdx.x + threadIdx.x;
    fx[my_idx] = 0.0;
    fy[my_idx] = 0.0;

    //printf("Calculate force. idx: %d\n", my_idx);
    for (int i = 0; i < n; ++i) {
        if (i == my_idx) continue;

        float dx = array_x[i] - array_x[my_idx];
        float dy = array_y[i] - array_y[my_idx];

        //printf("idx: %d, i: %d, dx: %f, dy: %f, xi: %f, yi: %f\n", my_idx, i, dx, dy, array_x[i], array_y[i]);
        
        float squared_dist = dx*dx + dy*dy;
        float dist = sqrtf(squared_dist);
        float force = G * masses[my_idx] * masses[i] / (squared_dist * dist);
        
        //printf("idx: %d, i: %d, force: %f", my_idx, i, force);

        fx[my_idx] += force * dx;
        fy[my_idx] += force * dy;
        //printf("idx: %d, i: %d, fx=%f, fy=%f\n", my_idx, i, fx[my_idx], fy[my_idx]);
    }

    update_points(fx, fy, masses, array_x, array_y, v_x, v_y, n, delta_t);
}

__host__ void freeMem(float *masses, float *array_x, float *array_y, float *vs_x, float *vs_y, float *fx, float *fy) {
    hipFree(masses);
    hipFree(array_x);
    hipFree(array_y);
    hipFree(vs_x);
    hipFree(vs_y);
    hipFree(fx);
    hipFree(fy);
}

void generate_bodies(float *masses, float *array_x, float *array_y, float *v_x, float *v_y, int n) {
    for(int i = 0; i < n; ++i) {
        masses[i] = ((float) rand()) / (RAND_MAX >> 10); 
        array_x[i] = 2.0 * ((float) rand()) / RAND_MAX - 1.0;
        array_y[i] = 2.0 * ((float) rand()) / RAND_MAX - 1.0;
        v_x[i] = 2.0 * ((float) rand()) / RAND_MAX - 1.0;
        v_y[i] = 2.0 * ((float) rand()) / RAND_MAX - 1.0;
        printf("Generating body: i=%d m=%f x=%f y=%f vx=%f vy=%f\n", i, masses[i], array_x[i], array_y[i], v_x[i], v_y[i]);
    }
}

void parse_csv(const char *filename, int n, float *m, float *x, float *y, float *vx, float *vy) {
    FILE *file = fopen(filename, "r");
    if (file == NULL) {
        perror("Failed to open file");
        exit(EXIT_FAILURE);
    }

    char line[MAX_LINE_LENGTH];
    int count = 0;

    printf("Parsing csv:\n");

    // Skip the first lines n and header
    fgets(line, sizeof(line), file);
    fgets(line, sizeof(line), file);
    
    while (fgets(line, sizeof(line), file) && count < n) {
        char *token = strtok(line, ";");
        if (token != NULL) {
            m[count] = atof(token);
            token = strtok(NULL, ";");
            x[count] = atof(token);
            token = strtok(NULL, ";");
            y[count] = atof(token);
            token = strtok(NULL, ";");
            vx[count] = atof(token);
            token = strtok(NULL, ";");
            vy[count] = atof(token);
            printf("%d : %f %f %f %f %f\n", count, m[count], x[count], y[count], vx[count], vy[count]);
            count++;
        }
    }

    fclose(file);
}

int main(int argc, char* argv[]) 
{
    printf("Start\n");
    
    int n; // кол-во тел и потоков
    float t_end = 100.0; // максимальный промежуток времени
    float time_step_count = 100.0;
    float delta_t = t_end / time_step_count;
    int block_cnt = 1;

    FILE *file = fopen("input.csv", "r");
    if (file == NULL) {
        perror("Failed to open file");
        return EXIT_FAILURE;
    }

    if (fscanf(file, "%d", &n) != 1) {
        fprintf(stderr, "Failed to read the value of n\n");
        fclose(file);
        return EXIT_FAILURE;
    }

    fclose(file);

    float *masses;
    float *array_x;
    float *array_y;
    float *vs_x;
    float *vs_y;
    float *fx;
    float *fy;

    hipMallocManaged(&masses, n*sizeof(float));
    hipMallocManaged(&array_x, n * sizeof(float));
    hipMallocManaged(&array_y, n * sizeof(float));
    hipMallocManaged(&vs_x, n * sizeof(float));
    hipMallocManaged(&vs_y, n * sizeof(float));
    hipMallocManaged(&fx, n * sizeof(float));
    hipMallocManaged(&fy, n * sizeof(float));

    parse_csv("input.csv", n, masses, array_x, array_y, vs_x, vs_y);

    float current_time = 0.0;
    while(current_time < t_end) {
        printf("%f ", current_time);
        for(int i = 0; i < n; ++i) {
            printf("%f %f ", array_x[i], array_y[i]);
        }
        printf("\n");
        calculate_force<<<block_cnt, n>>>(fx,  fy,  masses,  array_x,  array_y, vs_x, vs_y, n, delta_t);
        hipDeviceSynchronize();
        current_time += delta_t;
    }

    hipDeviceSynchronize();

    freeMem( masses, array_x, array_y, vs_x, vs_y, fx, fy);

    return 0;
}